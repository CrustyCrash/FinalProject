#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <iostream>


// Error checking macro
#define CHECK_CUDA_ERROR(call) {                                          \
    hipError_t err = call;                                               \
    if (err != hipSuccess) {                                             \
        std::cerr << "CUDA error in file '" << __FILE__                   \
                  << "' in line " << __LINE__ << ": "                     \
                  << hipGetErrorString(err) << " (" << err << ")"        \
                  << std::endl;                                           \
        hipDeviceReset();                                                \
        exit(EXIT_FAILURE);                                               \
    }                                                                     \
}

#define NSPEEDS         9
#define PARAMFILE       "input.params"
#define OBSTACLEFILE    "obstacles_300x200.dat"
#define FINALSTATEFILE  "final_state%s%s.dat"
#define AVVELSFILE      "av_vels%s%s.dat"

char finalStateFile[128];
char avVelocityFile[128];

/* struct to hold the parameter values */
typedef struct {
    int nx;            /* no. of cells in y-direction */
    int ny;            /* no. of cells in x-direction */
    int maxIters;      /* no. of iterations */
    int reynolds_dim;  /* dimension for Reynolds number */
    double density;    /* density per link */
    double accel;      /* density redistribution */
    double omega;      /* relaxation parameter */
} t_param;


/* struct to hold the 'speed' values */
typedef struct {
    double speeds[NSPEEDS];
} t_speed;

void die(const char *message, const int line, const char *file) {
    fprintf(stderr, "Error at line %d of file %s:\n", line, file);
    fprintf(stderr, "%s\n", message);
    fflush(stderr);
    exit(EXIT_FAILURE);
}


int initialise(t_param *params, t_speed **cells_ptr, t_speed **tmp_cells_ptr, int **obstacles_ptr) {
    FILE *fp;          /* file pointer */
    int ii, jj;        /* generic counters */
    int xx, yy;        /* generic array indices */
    int blocked;       /* indicates whether a cell is blocked by an obstacle */
    int retval;        /* to hold return value for checking */
    double w0, w1, w2; /* weighting factors */

    /* open the parameter file */
    fp = fopen(PARAMFILE, "r");
    if (fp == NULL) {
        die("could not open file input.params", __LINE__, __FILE__);
    }

    /* read in the parameter values */
    retval = fscanf(fp, "%d\n", &(params->nx));
    if (retval != 1)
        die("could not read param file: nx", __LINE__, __FILE__);
    retval = fscanf(fp, "%d\n", &(params->ny));
    if (retval != 1)
        die("could not read param file: ny", __LINE__, __FILE__);
    retval = fscanf(fp, "%d\n", &(params->maxIters));
    if (retval != 1)
        die("could not read param file: maxIters", __LINE__, __FILE__);
    retval = fscanf(fp, "%d\n", &(params->reynolds_dim));
    if (retval != 1)
        die("could not read param file: reynolds_dim", __LINE__, __FILE__);
    retval = fscanf(fp, "%lf\n", &(params->density));
    if (retval != 1)
        die("could not read param file: density", __LINE__, __FILE__);
    retval = fscanf(fp, "%lf\n", &(params->accel));
    if (retval != 1)
        die("could not read param file: accel", __LINE__, __FILE__);
    retval = fscanf(fp, "%lf\n", &(params->omega));
    if (retval != 1)
        die("could not read param file: omega", __LINE__, __FILE__);

    /* and close up the file */
    fclose(fp);

    *cells_ptr = (t_speed *) malloc(sizeof(t_speed) * (params->ny * params->nx));
    if (*cells_ptr == NULL)
        die("cannot allocate memory for cells", __LINE__, __FILE__);

    /* the map of obstacles */
    *obstacles_ptr = (int *)malloc(sizeof(int) * (params->ny * params->nx));
    if (*obstacles_ptr == NULL)
        die("cannot allocate column memory for obstacles", __LINE__, __FILE__);

    /* initialise densities */
    w0 = params->density * 4.0 / 9.0;
    w1 = params->density / 9.0;
    w2 = params->density / 36.0;

    for (ii = 0; ii < params->ny; ii++) {
        for (jj = 0; jj < params->nx; jj++) {
            /* centre */
            (*cells_ptr)[ii * params->nx + jj].speeds[0] = w0;
            /* axis directions */
            (*cells_ptr)[ii * params->nx + jj].speeds[1] = w1;
            (*cells_ptr)[ii * params->nx + jj].speeds[2] = w1;
            (*cells_ptr)[ii * params->nx + jj].speeds[3] = w1;
            (*cells_ptr)[ii * params->nx + jj].speeds[4] = w1;
            /* diagonals */
            (*cells_ptr)[ii * params->nx + jj].speeds[5] = w2;
            (*cells_ptr)[ii * params->nx + jj].speeds[6] = w2;
            (*cells_ptr)[ii * params->nx + jj].speeds[7] = w2;
            (*cells_ptr)[ii * params->nx + jj].speeds[8] = w2;
        }
    }

    /* first set all cells in obstacle array to zero */
    for (ii = 0; ii < params->ny; ii++) {
        for (jj = 0; jj < params->nx; jj++) {
            (*obstacles_ptr)[ii * params->nx + jj] = 0;
        }
    }

    /* open the obstacle data file */
    fp = fopen(OBSTACLEFILE, "r");
    if (fp == NULL) {
        die("could not open file obstacles", __LINE__, __FILE__);
    }

    /* read-in the blocked cells list */
    while ((retval = fscanf(fp, "%d %d %d\n", &xx, &yy, &blocked)) != EOF) {
        /* some checks */
        if (retval != 3)
            die("expected 3 values per line in obstacle file", __LINE__, __FILE__);
        if (xx < 0 || xx > params->nx - 1)
            die("obstacle x-coord out of range", __LINE__, __FILE__);
        if (yy < 0 || yy > params->ny - 1)
            die("obstacle y-coord out of range", __LINE__, __FILE__);
        if (blocked != 1)
            die("obstacle blocked value should be 1", __LINE__, __FILE__);
        /* assign to array */
        (*obstacles_ptr)[yy * params->nx + xx] = blocked;
    }

    /* and close the file */
    fclose(fp);

    return EXIT_SUCCESS;
}



__global__ void accelerate_flow(const t_param params, t_speed *cells, const int *obstacles)
{
    int ii = threadIdx.x + blockIdx.x * blockDim.x;
    int offset; /* generic counters */
    double *speeds;

    /* compute weighting factors */
    const double w1 = params.density * params.accel / 9.0;
    const double w2 = params.density * params.accel / 36.0;

    if (ii >= params.ny) return; // NEED TO REVIEW. Might change it to ii > params.ny

    offset = ii * params.nx /* + jj (where jj=0) */;
    speeds = cells[offset].speeds;
    /* if the cell is not occupied and we don't send a density negative */
    if (!obstacles[offset] && (speeds[3] - w1) > 0.0 && (speeds[6] - w2) > 0.0 && (speeds[7] - w2) > 0.0) {
        /* increase 'east-side' densities */
        speeds[1] += w1;
        speeds[5] += w2;
        speeds[8] += w2;
        /* decrease 'west-side' densities */
        speeds[3] -= w1;
        speeds[6] -= w2;
        speeds[7] -= w2;
    }
}

__global__ void collision(const t_param params, t_speed *src_cells, t_speed *dst_cells, const int *obstacles)
{
    int ii = blockIdx.y * blockDim.y + threadIdx.y;
    int jj = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = ii * params.nx + jj;

    if (ii >= params.ny || jj >= params.nx) return;

    int kk;
    const double w0 = 4.0 / 9.0;       /* weighting factor */
    const double w1 = 1.0 / 9.0;       /* weighting factor */
    const double w2 = 1.0 / 36.0;      /* weighting factor */
    double u_x, u_y;                   /* av. velocities in x and y directions */
    double u[NSPEEDS];                 /* directional velocities */
    double d_equ[NSPEEDS];             /* equilibrium densities */
    double u_sq;                       /* squared velocity */
    double local_density;              /* sum of densities in a particular cell */
    int x_e, x_w, y_n, y_s;            /* indices of neighbouring cells */
    double speeds[NSPEEDS];

    double *dst_speeds = dst_cells[offset].speeds;

    /* PROPAGATE: determine indices of axis-direction neighbours respecting periodic boundary conditions (wrap around) */
        y_s = (ii + 1) % params.ny;
        x_w = (jj + 1) % params.nx;
        y_n = (ii == 0) ? (ii + params.ny - 1) : (ii - 1);
        x_e = (jj == 0) ? (jj + params.nx - 1) : (jj - 1);

        /* if the cell contains an obstacle */
        if (obstacles[ii * params.nx + jj])
        {
            /* PROPAGATE ANB REBOUND: propagate and mirror densities from neighbouring cells into the current cell. */
            dst_speeds[0] = src_cells[ii * params.nx + jj].speeds[0];  /* central cell, no movement */
            dst_speeds[1] = src_cells[ii * params.nx + x_w].speeds[3];  /* west */
            dst_speeds[2] = src_cells[y_s * params.nx + jj].speeds[4];  /* south */
            dst_speeds[3] = src_cells[ii * params.nx + x_e].speeds[1];  /* east */
            dst_speeds[4] = src_cells[y_n * params.nx + jj].speeds[2];  /* north */
            dst_speeds[5] = src_cells[y_s * params.nx + x_w].speeds[7];  /* south-west */
            dst_speeds[6] = src_cells[y_s * params.nx + x_e].speeds[8];  /* south-east */
            dst_speeds[7] = src_cells[y_n * params.nx + x_e].speeds[5];  /* north-east */
            dst_speeds[8] = src_cells[y_n * params.nx + x_w].speeds[6];  /* north-west */
        }

            else
            {
                /* PROPAGATE: propagate densities from neighbouring cells into the current cells, following appropriate directions of
                 * travel and writing into a temporary buffer.
                 */
                speeds[0] = src_cells[ii * params.nx + jj].speeds[0];  /* central cell, no movement */
                speeds[1] = src_cells[ii * params.nx + x_e].speeds[1];  /* east */
                speeds[2] = src_cells[y_n * params.nx + jj].speeds[2];  /* north */
                speeds[3] = src_cells[ii * params.nx + x_w].speeds[3];  /* west */
                speeds[4] = src_cells[y_s * params.nx + jj].speeds[4];  /* south */
                speeds[5] = src_cells[y_n * params.nx + x_e].speeds[5];  /* north-east */
                speeds[6] = src_cells[y_n * params.nx + x_w].speeds[6];  /* north-west */
                speeds[7] = src_cells[y_s * params.nx + x_w].speeds[7];  /* south-west */
                speeds[8] = src_cells[y_s * params.nx + x_e].speeds[8];  /* south-east */

                /* COLLISION */
                /* compute local density total */
                local_density = speeds[0] + speeds[1] + speeds[2] + speeds[3] + speeds[4] + speeds[5] + speeds[6] + speeds[7] + speeds[8];
                
                /* compute x velocity component */
                u_x = (speeds[1] + speeds[5] + speeds[8] - (speeds[3] + speeds[6] + speeds[7])) / local_density;
                
                /* compute y velocity component */
                u_y = (speeds[2] + speeds[5] + speeds[6] - (speeds[4] + speeds[7] + speeds[8])) / local_density;
               
                /* directional velocity components */
                u[1] = u_x;       /* east */
                u[2] = u_y;       /* north */
                u[5] = u_x + u_y; /* north-east */
                u[6] = -u_x + u_y; /* north-west */

                /* velocity squared over twice the speed of sound */
                u_sq = (u_x * u_x + u_y * u_y) * 1.5;
                /* equilibrium densities */
                /* zero velocity density: weight w0 */
                d_equ[0] = w0 * local_density * (1.0 - u_sq);

                /* axis speeds: weight w1 */
                d_equ[1] = w1 * local_density * (1.0 + u[1] * 3.0 + (u[1] * u[1] * 4.5) - u_sq);
                d_equ[2] = w1 * local_density * (1.0 + u[2] * 3.0 + (u[2] * u[2] * 4.5) - u_sq);
                d_equ[3] = w1 * local_density * (1.0 - u[1] * 3.0 + (u[1] * u[1] * 4.5) - u_sq);
                d_equ[4] = w1 * local_density * (1.0 - u[2] * 3.0 + (u[2] * u[2] * 4.5) - u_sq);
                /* diagonal speeds: weight w2 */
                d_equ[5] = w2 * local_density * (1.0 + u[5] * 3.0 + (u[5] * u[5] * 4.5) - u_sq);
                d_equ[6] = w2 * local_density * (1.0 + u[6] * 3.0 + (u[6] * u[6] * 4.5) - u_sq);
                d_equ[7] = w2 * local_density * (1.0 - u[5] * 3.0 + (u[5] * u[5] * 4.5) - u_sq);
                d_equ[8] = w2 * local_density * (1.0 - u[6] * 3.0 + (u[6] * u[6] * 4.5) - u_sq);

                /* relaxation step */
                for (kk = 0; kk < NSPEEDS; kk++) {
                    speeds[kk] += params.omega * (d_equ[kk] - speeds[kk]);
                }
                *((t_speed *) dst_speeds) = *((t_speed *) speeds);

            }
}


__global__ void compute_av_velocity_kernel(const t_param params, t_speed* d_cells, int* d_obstacles, double* d_tot_u_x, int* d_tot_cells) {
    int ii = blockIdx.y * blockDim.y + threadIdx.y;
    int jj = blockIdx.x * blockDim.x + threadIdx.x;

    if (ii < params.ny && jj < params.nx) {
        int offset = ii * params.nx + jj;

        if (!d_obstacles[offset]) {
            t_speed cell = d_cells[offset];
            double local_density = cell.speeds[0] + cell.speeds[1] + cell.speeds[2] + cell.speeds[3] + cell.speeds[4] + cell.speeds[5] + cell.speeds[6] + cell.speeds[7] + cell.speeds[8];
            double u_x = (cell.speeds[1] + cell.speeds[5] + cell.speeds[8] - (cell.speeds[3] + cell.speeds[6] + cell.speeds[7])) / local_density;

            atomicAdd(d_tot_u_x, u_x);
            atomicAdd(d_tot_cells, 1);
        }
    }
}

double av_velocity(const t_param params, t_speed* cells, int* obstacles) {
    t_speed* d_cells;
    int* d_obstacles;
    double* d_tot_u_x;
    int* d_tot_cells;

    size_t cells_size = params.nx * params.ny * sizeof(t_speed);
    size_t obstacles_size = params.nx * params.ny * sizeof(int);

    // Allocate device memory
    hipMalloc(&d_cells, cells_size);
    hipMalloc(&d_obstacles, obstacles_size);
    hipMalloc(&d_tot_u_x, sizeof(double));
    hipMalloc(&d_tot_cells, sizeof(int));

    // Initialize device memory
    hipMemset(d_tot_u_x, 0, sizeof(double));
    hipMemset(d_tot_cells, 0, sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_cells, cells, cells_size, hipMemcpyHostToDevice);
    hipMemcpy(d_obstacles, obstacles, obstacles_size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((params.nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (params.ny + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch kernel
    compute_av_velocity_kernel<<<numBlocks, threadsPerBlock>>>(params, d_cells, d_obstacles, d_tot_u_x, d_tot_cells);

    // Copy results back to host
    double tot_u_x;
    int tot_cells;
    hipMemcpy(&tot_u_x, d_tot_u_x, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&tot_cells, d_tot_cells, sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_cells);
    hipFree(d_obstacles);
    hipFree(d_tot_u_x);
    hipFree(d_tot_cells);

    return tot_cells > 0 ? tot_u_x / (double) tot_cells : 0.0;
}


__host__ double calculate_reynolds_number(const t_param params, double avg_velocity) {
    // The formula for Reynolds number
    double nu = 1.0 / 6.0 * (2.0 / params.omega - 1.0);  // Kinematic viscosity
    return (avg_velocity * params.reynolds_dim) / nu;
}


int main(int argc, char* argv[])
{
    t_param params;            /* struct to hold parameter values */
    //src_cells needs cudaMemcpy
    t_speed *src_cells = NULL; /* source grid containing fluid densities */
    //dst and tmp needs cudaMalloc
    t_speed *dst_cells = NULL; /* destination grid containing fluid densities */
    t_speed *temp_swap = NULL; /* temporary cell pointer variable used to swap source and destination grid pointers */
    // obstacles needs cudaMemcpy
    int *obstacles = NULL;     /* grid indicating which cells are blocked */
    // av_vels needs cudaMalloc
    double av_vels = 0.0; 
    int ii;                    /* generic counter */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;


    if (argc > 1) {
        sprintf(finalStateFile, FINALSTATEFILE, ".", argv[1]);
        sprintf(avVelocityFile, AVVELSFILE, ".", argv[1]);
    } else {
        sprintf(finalStateFile, FINALSTATEFILE, "", "");
        sprintf(avVelocityFile, AVVELSFILE, "", "");
    }

    /* initialise our data structures and load values from file */
    initialise(&params, &src_cells, &dst_cells, &obstacles);

    t_param* device_params;
    CHECK_CUDA_ERROR(hipMalloc(&device_params, sizeof(t_param)));
    CHECK_CUDA_ERROR(hipMemcpy(device_params, &params, sizeof(t_param), hipMemcpyHostToDevice));

    t_speed* device_src_cells;
    CHECK_CUDA_ERROR(hipMalloc((void**)&device_src_cells, sizeof(t_speed) * (params.ny * params.nx)));
    CHECK_CUDA_ERROR(hipMemcpy(device_src_cells, src_cells, sizeof(t_speed) * (params.ny * params.nx), hipMemcpyHostToDevice));

    t_speed* device_dst_cells;
    CHECK_CUDA_ERROR(hipMalloc((void**)&device_dst_cells, sizeof(t_speed) * (params.ny * params.nx)));
    
    int* device_obstacles;
    CHECK_CUDA_ERROR(hipMalloc((void**)&device_obstacles, sizeof(int) * (params.ny * params.nx)));
    CHECK_CUDA_ERROR(hipMemcpy(device_obstacles, obstacles, sizeof(int) * (params.ny * params.nx), hipMemcpyHostToDevice));

    t_speed* device_temp_swap;
    CHECK_CUDA_ERROR(hipMalloc((void**)&device_temp_swap, sizeof(t_speed)));

    dim3 blockDim(16);
    dim3 gridDim((params.ny + blockDim.y - 1) / blockDim.y);

    dim3 blockDim2(16,16);
    dim3 gridDim2((params.nx + blockDim.x - 1) / blockDim.x, (params.ny + blockDim.y - 1) / blockDim.y);

    hipEventRecord(start);

    for (ii = 0; ii < params.maxIters; ii++)
    {
        accelerate_flow<<<gridDim,blockDim>>>(params, device_src_cells, device_obstacles);
        hipDeviceSynchronize();
        collision<<<gridDim2,blockDim2>>>(params, device_src_cells, device_dst_cells, device_obstacles);
        hipDeviceSynchronize();
        temp_swap = device_src_cells;
        device_src_cells = device_dst_cells;
        device_dst_cells = temp_swap;
    }

    CHECK_CUDA_ERROR(hipMemcpy(src_cells, device_src_cells, sizeof(t_speed) * (params.ny * params.nx), hipMemcpyDeviceToHost));
    
    av_vels = av_velocity(params, src_cells, obstacles);

    // Record the stop event
    hipEventRecord(stop);

    // Wait for the stop event to complete
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);

    double reynolds_number = calculate_reynolds_number(params, av_vels);
    printf("Reynolds number:\t%.12E\n", reynolds_number);
    printf("Time elapsed: %f ms\n", milliseconds);

}